#include "hip/hip_runtime.h"
#define KERNELS_INCLUDE_IMPLEMENTATION
#include "kernels.h"

using SubgridType = t8gpu::Subgrid<4, 4, 4>;

// We explicitely instantiate the kernel functions for the 3D subgrid
// in a separate compilation unit for better compilation speed
// (instead of having one big translation unit, we split it up in 3:
// main_3d.cu, solver_3d.cu and kernels_3d.cu).
template __global__ void compute_inner_fluxes<SubgridType>(
    t8gpu::SubgridMemoryAccessorOwn<VariableList, SubgridType>              variables,
    t8gpu::SubgridMemoryAccessorOwn<VariableList, SubgridType>              fluxes,
    typename SubgridCompressibleEulerSolver<SubgridType>::float_type const* volumes);

template __global__ void compute_outer_fluxes<SubgridType>(
    typename t8gpu::SubgridMeshConnectivityAccessor<typename SubgridCompressibleEulerSolver<SubgridType>::float_type,
                                                    SubgridType> connectivity,
    t8gpu::SubgridMemoryAccessorAll<VariableList, SubgridType>   variables,
    t8gpu::SubgridMemoryAccessorAll<VariableList, SubgridType>   fluxes);

template __global__ void compute_refinement_criteria<SubgridType>(
    typename SubgridType::Accessor<typename SubgridCompressibleEulerSolver<SubgridType>::float_type> density,
    typename SubgridCompressibleEulerSolver<SubgridType>::float_type*       refinement_criteria,
    typename SubgridCompressibleEulerSolver<SubgridType>::float_type const* volumes,
    t8_locidx_t                                                             num_local_elements);
