#include "hip/hip_runtime.h"
#include <advection_solver.h>

#include <cassert>
#include <cmath>
#include <iostream>

#include <t8.h>
#include <t8_cmesh.h>
#include <t8_cmesh/t8_cmesh_examples.h>

#include <t8_forest/t8_forest.h>
#include <t8_schemes/t8_default/t8_default_cxx.hxx>

int adapt_callback(t8_forest_t forest, t8_forest_t forest_from, t8_locidx_t which_tree, t8_locidx_t lelement_id,
		   t8_eclass_scheme_c* ts, const int is_family, const int num_elements, t8_element_t* elements[]) {


  double center[] = {0.5, 0.5, 0.0};
  double element_centroid[3];
  t8_forest_element_centroid (forest_from, which_tree, elements[0], element_centroid);

  double distance_squared = (center[0]-element_centroid[0])*(center[0]-element_centroid[0]) + (center[1]-element_centroid[1])*(center[1]-element_centroid[1]);

  if (distance_squared < 0.2*0.2) {
    return 1;
  } else if (is_family) {

    double centroid[] = {0.0, 0.0, 0.0};
    double current_element_centroid[] = {0.0, 0.0, 0.0};
    for (size_t i=0; i<4; i++) {
      t8_forest_element_centroid (forest_from, which_tree, elements[i], current_element_centroid);
      for (size_t j=0; j<3; j++) {
	centroid[j] += current_element_centroid[j] / 4.0;
      }
    }
    double distance_squared = (center[0]-centroid[0])*(center[0]-centroid[0]) + (center[1]-centroid[1])*(center[1]-centroid[1]);
    return (distance_squared > 0.4*0.4) ? -1 : 0;
  } else {
    return 0;
  }
}

advection_solver_t::advection_solver_t() : comm(sc_MPI_COMM_WORLD),
					   cmesh(t8_cmesh_new_periodic(comm, dim)),
					   scheme(t8_scheme_new_default_cxx()),
					   forest(t8_forest_new_uniform(cmesh, scheme, level, false, comm)),
					   element_variable(t8_forest_get_local_num_elements(forest)),
					   element_volume(t8_forest_get_local_num_elements(forest)),
					   delta_t(1.0*std::pow(0.5, level+1) / sqrt(2.0)) {

  t8_forest_t new_forest = t8_forest_new_adapt(forest, adapt_callback, 0, 0, nullptr);
  forest = new_forest;

  element_variable.resize(t8_forest_get_local_num_elements(forest));
  element_volume.resize(t8_forest_get_local_num_elements(forest));

  t8_locidx_t num_local_elements = element_variable.size();

  t8_locidx_t num_local_trees = t8_forest_get_num_local_trees(forest);
  t8_locidx_t element_idx = 0;
  for (t8_locidx_t tree_idx = 0; tree_idx < num_local_trees; tree_idx++) {
    t8_eclass_t tree_class = t8_forest_get_tree_class (forest, tree_idx);
    t8_eclass_scheme_c *eclass_scheme = t8_forest_get_eclass_scheme (forest, tree_class);

    t8_locidx_t num_elements_in_tree = t8_forest_get_tree_num_elements(forest, tree_idx);
    for (t8_locidx_t tree_element_idx = 0; tree_element_idx < num_elements_in_tree; tree_element_idx++) {
      const t8_element_t* element = t8_forest_get_element_in_tree(forest, tree_idx, tree_element_idx);

      double center[3];
      t8_forest_element_centroid(forest, tree_idx, element, center);

      element_variable[element_idx] = (0.5-center[0])*(0.5-center[0]) + (0.5-center[1])*(0.5-center[1]) + (0.5-center[2])*(0.5-center[2]);
      element_volume[element_idx] = t8_forest_element_volume(forest, tree_idx, element);

      size_t num_faces = static_cast<size_t>(eclass_scheme->t8_element_num_faces(element));
      for (size_t face_idx=0; face_idx < num_faces; face_idx++) {
	int num_neighbors;
	int* dual_faces;
	t8_locidx_t* neighbor_ids;
	t8_element_t** neighbors;
	t8_eclass_scheme_c* neigh_scheme;

	t8_forest_leaf_face_neighbors(forest, tree_idx, element, &neighbors, face_idx, &dual_faces, &num_neighbors,
				      &neighbor_ids, &neigh_scheme, 1);

	if ((num_neighbors == 1) && ((neighbor_ids[0] > element_idx) ||
				     (neighbor_ids[0] < element_idx  && neigh_scheme[0].t8_element_level(neighbors[0]) < eclass_scheme->t8_element_level(element)
				      ))) {
	  face_neighbors.push_back(std::array<int,2>{element_idx, neighbor_ids[0]});
	  double face_normal[3];
	  t8_forest_element_face_normal(forest, tree_idx, element, face_idx, face_normal);
	  face_normals.push_back(std::array<double,2>{face_normal[0], face_normal[1]});
	  face_area.push_back(t8_forest_element_face_area(forest, tree_idx, element, face_idx));
	}

	T8_FREE(neighbors);
	T8_FREE(dual_faces);
	T8_FREE(neighbor_ids);
      }

      element_idx++;
    }
  }

  hipMalloc(&device_element_variable_prev, sizeof(double)*num_local_elements);
  hipMalloc(&device_element_variable_next, sizeof(double)*num_local_elements);
  hipMalloc(&device_element_fluxes, sizeof(double)*num_local_elements);
  hipMalloc(&device_element_volume, sizeof(double)*num_local_elements);

  hipMalloc(&device_face_neighbors, sizeof(int)*face_neighbors.size()*2);
  hipMalloc(&device_face_normals, sizeof(double)*face_normals.size()*2);
  hipMalloc(&device_face_area, sizeof(double)*face_normals.size()*2);

  hipMemcpy(device_element_variable_next, element_variable.data(), element_variable.size()*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(device_element_volume, element_volume.data(), element_volume.size()*sizeof(double), hipMemcpyHostToDevice);
  hipMemset(device_element_fluxes, 0, element_variable.size());

  hipMemcpy(device_face_neighbors, face_neighbors.data(), face_neighbors.size()*2*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(device_face_normals, face_normals.data(), face_normals.size()*2*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(device_face_area, face_area.data(), face_area.size()*2*sizeof(double), hipMemcpyHostToDevice);
}

advection_solver_t::~advection_solver_t() {
  t8_forest_unref(&forest);
  t8_cmesh_destroy(&cmesh);

  hipFree(device_element_variable_prev);
  hipFree(device_element_variable_next);
  hipFree(device_element_fluxes);
  hipFree(device_element_volume);

  hipFree(device_face_neighbors);
  hipFree(device_face_normals);
  hipFree(device_face_area);
}

__global__ static void compute_fluxes(double const* __restrict__ variable,
				      double* __restrict__ fluxes,
				      double const* __restrict__ normal,
				      double const* __restrict__ area,
				      int const* e_idx) {

  double a[2] = {0.5*sqrt(2.0), 0.5*sqrt(2.0)};

  int i = blockIdx.x * blockDim.x + threadIdx.x;

  double flux = area[i]*(a[0]*normal[2*i]+a[1]*normal[2*i+1]);

  if (flux > 0.0) {
    flux *= variable[e_idx[2*i]];
  } else {
    flux *= variable[e_idx[2*i+1]];
  }

  atomicAdd(&fluxes[e_idx[2*i]], -flux);
  atomicAdd(&fluxes[e_idx[2*i+1]], flux);
}

__global__ static void explicit_euler_time_step(double const* __restrict__ variable_prev,
						double* __restrict__ variable_next,
						double const * __restrict__ volume,
						double* __restrict__ fluxes,
						double delta_t) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  variable_next[i] = variable_prev[i] + delta_t/volume[i]*fluxes[i];

  fluxes[i] = 0.0;
}

void advection_solver_t::iterate() {
  std::swap(device_element_variable_next, device_element_variable_prev);

  compute_fluxes<<<face_area.size(), 1>>>(device_element_variable_prev,
					     device_element_fluxes,
					     device_face_normals,
					     device_face_area,
					     device_face_neighbors);

  explicit_euler_time_step<<<element_volume.size(),1>>>(device_element_variable_prev,
							device_element_variable_next,
							device_element_volume,
							device_element_fluxes,
							delta_t);
}

void advection_solver_t::save_vtk(const std::string& prefix) {
  hipMemcpy(element_variable.data(), device_element_variable_next, element_variable.size()*sizeof(double), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  t8_vtk_data_field_t vtk_data_field =  {};
  vtk_data_field.type = T8_VTK_SCALAR;
  strcpy(vtk_data_field.description, "diffusion variable");
  vtk_data_field.data = element_variable.data();

  t8_forest_write_vtk_ext(forest, prefix.c_str(), 1, 1, 1, 1, 0, 0, 0, 1, &vtk_data_field);
}
