#include "hip/hip_runtime.h"
#include <advection_solver.h>
#include <t8.h>
#include <t8_cmesh.h>
#include <t8_cmesh/t8_cmesh_examples.h>
#include <t8_forest/t8_forest.h>
#include <t8_forest/t8_forest_iterate.h>
#include <t8_forest/t8_forest_partition.h>
#include <utils/hip/hip_runtime.h>
#include <utils/profiling.h>

#include <cassert>
#include <cmath>
#include <iostream>
#include <t8_schemes/t8_default/t8_default_cxx.hxx>

struct forest_user_data_t {
  thrust::host_vector<double>* element_refinement_criteria;
};

static int adapt_callback_initialization(t8_forest_t forest, t8_forest_t forest_from, t8_locidx_t which_tree, t8_locidx_t lelement_id,
					 t8_eclass_scheme_c* ts, const int is_family, const int num_elements, t8_element_t* elements[]);

static int adapt_callback_iteration(t8_forest_t forest, t8_forest_t forest_from, t8_locidx_t which_tree, t8_locidx_t lelement_id, t8_eclass_scheme_c* ts,
				    const int is_family, const int num_elements, t8_element_t* elements[]);

__global__ static void compute_refinement_criteria(double const* __restrict__ variable, double* __restrict__ criteria, int nb_elements);

__global__ static void adapt_variable_and_volume(double const* __restrict__ variable_old, double const* __restrict__ volume_old,
                                                 double* __restrict__ variable_new, double* __restrict__ volume_new, t8_locidx_t* adapt_data,
                                                 int nb_new_elements);

__global__ void partition_data(int* __restrict__ ranks, t8_locidx_t* __restrict__ indices,
			       double* __restrict__ new_variable, double* __restrict__ new_volume,
			       double const*const* __restrict__ old_variable, double const*const* __restrict__ old_volume,
			       int num_new_elements);

__global__ static void compute_fluxes(double** __restrict__ variables, double** __restrict__ fluxes, double const* __restrict__ normal,
                                      double const* __restrict__ area, int const* e_idx, int* rank, t8_locidx_t* indices, int nb_edges);

__global__ static void explicit_euler_time_step(double const* __restrict__ variable_prev, double* __restrict__ variable_next,
                                                double const* __restrict__ volume, double* __restrict__ fluxes, double delta_t, int nb_elements);

t8gpu::AdvectionSolver::AdvectionSolver(sc_MPI_Comm comm)
    : comm(comm),
      cmesh(t8_cmesh_new_periodic_hybrid(comm)),
      scheme(t8_scheme_new_default_cxx()),
      forest(t8_forest_new_uniform(cmesh, scheme, 6, true, comm)),
      delta_t(0.5 * std::pow(0.5, max_level) / sqrt(2.0)) {
  t8_forest_t new_forest {};
  t8_forest_init(&new_forest);
  t8_forest_set_adapt(new_forest, forest, adapt_callback_initialization, true);
  t8_forest_set_ghost(new_forest, true, T8_GHOST_FACES);
  t8_forest_set_balance(new_forest, forest, false);
  t8_forest_set_partition(new_forest, forest, true);
  t8_forest_commit(new_forest);
  forest = new_forest;

  MPI_Comm_size(comm, &nb_ranks);
  MPI_Comm_rank(comm, &rank);

  num_ghost_elements = t8_forest_get_num_ghosts(forest);
  num_local_elements = t8_forest_get_local_num_elements(forest);

  ranks.resize(num_local_elements + num_ghost_elements);
  indices.resize(num_local_elements + num_ghost_elements);
  for (t8_locidx_t i=0; i<num_local_elements; i++) {
    ranks[i] = rank;
    indices[i] = i;
  }
  sc_array* sc_array_ranks_wrapper {sc_array_new_data(ranks.data(), sizeof(int), num_local_elements + num_ghost_elements)};
  t8_forest_ghost_exchange_data(forest, sc_array_ranks_wrapper);
  sc_array_destroy(sc_array_ranks_wrapper);

  sc_array* sc_array_indices_wrapper {sc_array_new_data(indices.data(), sizeof(t8_locidx_t), num_local_elements + num_ghost_elements)};
  t8_forest_ghost_exchange_data(forest, sc_array_indices_wrapper);
  sc_array_destroy(sc_array_indices_wrapper);

  device_ranks = ranks;
  device_indices = indices;

  thrust::host_vector<double> element_variable(num_local_elements);
  thrust::host_vector<double> element_volume(num_local_elements);
  element_refinement_criteria.resize(num_local_elements);

  t8_locidx_t num_local_trees = t8_forest_get_num_local_trees(forest);
  t8_locidx_t element_idx = 0;
  for (t8_locidx_t tree_idx = 0; tree_idx < num_local_trees; tree_idx++) {
    t8_eclass_t tree_class {t8_forest_get_tree_class(forest, tree_idx)};
    t8_eclass_scheme_c* eclass_scheme {t8_forest_get_eclass_scheme(forest, tree_class)};

    t8_locidx_t num_elements_in_tree {t8_forest_get_tree_num_elements(forest, tree_idx)};
    for (t8_locidx_t tree_element_idx = 0; tree_element_idx < num_elements_in_tree; tree_element_idx++) {
      const t8_element_t* element {t8_forest_get_element_in_tree(forest, tree_idx, tree_element_idx)};

      double center[3];
      t8_forest_element_centroid(forest, tree_idx, element, center);

      element_variable[element_idx] = sqrt((0.5 - center[0]) * (0.5 - center[0]) + (0.5 - center[1]) * (0.5 - center[1])) - 0.25;
      element_volume[element_idx] = t8_forest_element_volume(forest, tree_idx, element);

      element_idx++;
    }
  }

  device_element_variable_prev.resize(num_local_elements);
  device_element_fluxes.resize(num_local_elements);
  device_element_refinement_criteria.resize(num_local_elements);

  device_element_variable_next = element_variable;
  device_element_volume = element_volume;
  double* device_element_fluxes_ptr {device_element_fluxes.get_own()};
  T8GPU_CUDA_CHECK_ERROR(hipMemset(device_element_fluxes_ptr, 0, sizeof(double)*num_local_elements));

  compute_edge_connectivity();
  device_face_neighbors = face_neighbors;
  device_face_normals = face_normals;
  device_face_area = face_area;

  // TODO: remove allocation out of RAII paradigm
  forest_user_data_t* forest_user_data = static_cast<forest_user_data_t*>(malloc(sizeof(forest_user_data_t)));
  assert(forest_user_data != nullptr);

  forest_user_data->element_refinement_criteria = &element_refinement_criteria;
  t8_forest_set_user_data(forest, forest_user_data);
}

t8gpu::AdvectionSolver::~AdvectionSolver() {
  forest_user_data_t* forest_user_data {static_cast<forest_user_data_t*>(t8_forest_get_user_data(forest))};
  free(forest_user_data);

  t8_forest_unref(&forest);
  t8_cmesh_destroy(&cmesh);
}

void t8gpu::AdvectionSolver::iterate() {
  std::swap(device_element_variable_next, device_element_variable_prev);

  double* device_element_fluxes_ptr {device_element_fluxes.get_own()};
  T8GPU_CUDA_CHECK_ERROR(hipMemset(device_element_fluxes_ptr, 0, sizeof(double)*device_element_fluxes.size()));

  hipDeviceSynchronize();
  MPI_Barrier(comm);
  constexpr int thread_block_size = 256;
  const int fluxes_num_blocks = (num_local_faces + thread_block_size - 1) / thread_block_size;
  compute_fluxes<<<fluxes_num_blocks, thread_block_size>>>(
							   device_element_variable_prev.get_all(),
							   device_element_fluxes.get_all(),
							   thrust::raw_pointer_cast(device_face_normals.data()),
							   thrust::raw_pointer_cast(device_face_area.data()),
							   thrust::raw_pointer_cast(device_face_neighbors.data()),
							   thrust::raw_pointer_cast(device_ranks.data()),
							   thrust::raw_pointer_cast(device_indices.data()),
							   num_local_faces);
  T8GPU_CUDA_CHECK_LAST_ERROR();
  hipDeviceSynchronize();
  MPI_Barrier(comm);

  const int euler_num_blocks = (t8_forest_get_local_num_elements(forest) + thread_block_size - 1) / thread_block_size;
  explicit_euler_time_step<<<euler_num_blocks, thread_block_size>>>(
      device_element_variable_prev.get_own(), device_element_variable_next.get_own(),
      device_element_volume.get_own(), device_element_fluxes.get_own(), delta_t, num_local_elements);
  T8GPU_CUDA_CHECK_LAST_ERROR();
}

void t8gpu::AdvectionSolver::adapt() {
  constexpr int thread_block_size = 256;
  const int fluxes_num_blocks = (num_local_elements + thread_block_size - 1) / thread_block_size;
  compute_refinement_criteria<<<fluxes_num_blocks, thread_block_size>>>(device_element_variable_next.get_own(),
                                                                        thrust::raw_pointer_cast(device_element_refinement_criteria.data()),
                                                                        num_local_elements);
  T8GPU_CUDA_CHECK_LAST_ERROR();

  element_refinement_criteria = device_element_refinement_criteria;

  t8_forest_ref(forest);
  assert(t8_forest_is_committed(forest));

  t8_forest_t adapted_forest {};
  t8_forest_init(&adapted_forest);
  t8_forest_set_adapt(adapted_forest, forest, adapt_callback_iteration, false);
  t8_forest_set_ghost(adapted_forest, true, T8_GHOST_FACES);
  t8_forest_set_balance(adapted_forest, forest, true);
  t8_forest_commit(adapted_forest);

  t8_locidx_t old_idx = 0;
  t8_locidx_t new_idx = 0;

  t8_locidx_t num_new_elements {t8_forest_get_local_num_elements(adapted_forest)};
  t8_locidx_t num_old_elements {t8_forest_get_local_num_elements(forest)};

  thrust::host_vector<double> adapted_element_variable(num_new_elements);
  thrust::host_vector<double> adapted_element_volume(num_new_elements);
  thrust::host_vector<t8_locidx_t> element_adapt_data(num_new_elements + 1);

  thrust::host_vector<t8_locidx_t> old_levels(num_old_elements);
  thrust::host_vector<t8_locidx_t> new_levels(num_new_elements);

  t8_locidx_t num_old_local_trees = {t8_forest_get_num_local_trees(forest)};
  t8_locidx_t num_new_local_trees = {t8_forest_get_num_local_trees(forest)};

  t8_locidx_t current_idx = 0;
  for (t8_locidx_t tree_idx = 0; tree_idx < num_old_local_trees; tree_idx++) {
    t8_eclass_t old_tree_class {t8_forest_get_tree_class(forest, tree_idx)};
    t8_eclass_scheme_c* old_scheme = {t8_forest_get_eclass_scheme(forest, old_tree_class)};

    t8_locidx_t num_elements_in_tree {t8_forest_get_tree_num_elements(forest, tree_idx)};

    for (t8_locidx_t elem_idx = 0; elem_idx < num_elements_in_tree; elem_idx++) {
      t8_element_t const* element {t8_forest_get_element_in_tree(forest, tree_idx, elem_idx)};
      old_levels[current_idx] = old_scheme->t8_element_level(element);
      current_idx++;
    }
  }

  current_idx = 0;
  for (t8_locidx_t tree_idx = 0; tree_idx < num_new_local_trees; tree_idx++) {
    t8_eclass_t new_tree_class {t8_forest_get_tree_class(adapted_forest, tree_idx)};
    t8_eclass_scheme_c* new_scheme = {t8_forest_get_eclass_scheme(adapted_forest, new_tree_class)};

    t8_locidx_t num_elements_in_tree {t8_forest_get_tree_num_elements(adapted_forest, tree_idx)};

    for (t8_locidx_t elem_idx = 0; elem_idx < num_elements_in_tree; elem_idx++) {
      t8_element_t const* element {t8_forest_get_element_in_tree(adapted_forest, tree_idx, elem_idx)};
      new_levels[current_idx] = new_scheme->t8_element_level(element);
      current_idx++;
    }
  }

  while (old_idx < num_old_elements && new_idx < num_new_elements) {
    int old_level = old_levels[old_idx];
    int new_level = new_levels[new_idx];

    if (old_level < new_level) {  // refined
      for (size_t i = 0; i < 4; i++) {
        element_adapt_data[new_idx + i] = old_idx;
      }
      old_idx += 1;
      new_idx += 4;
    } else if (old_level > new_level) {  // coarsened
      for (size_t i = 0; i < 4; i++) {
      }
      element_adapt_data[new_idx] = old_idx;
      old_idx += 4;
      new_idx += 1;
    } else {
      element_adapt_data[new_idx] = old_idx;
      old_idx += 1;
      new_idx += 1;
    }
  }
  element_adapt_data[new_idx] = old_idx;

  element_refinement_criteria.resize(num_new_elements);

  forest_user_data_t* forest_user_data {static_cast<forest_user_data_t*>(t8_forest_get_user_data(forest))};
  assert(forest_user_data != nullptr);

  t8_forest_set_user_data(adapted_forest, forest_user_data);
  t8_forest_unref(&forest);

  thrust::device_vector<double> device_element_variable_next_adapted(num_new_elements);
  thrust::device_vector<double> device_element_volume_adapted(num_new_elements);
  t8_locidx_t* device_element_adapt_data {};
  T8GPU_CUDA_CHECK_ERROR(hipMalloc(&device_element_adapt_data, (num_new_elements + 1) * sizeof(t8_locidx_t)));
  T8GPU_CUDA_CHECK_ERROR(
      hipMemcpy(device_element_adapt_data, element_adapt_data.data(), element_adapt_data.size() * sizeof(t8_locidx_t), hipMemcpyHostToDevice));
  const int adapt_num_blocks = (num_new_elements + thread_block_size - 1) / thread_block_size;
  adapt_variable_and_volume<<<adapt_num_blocks, thread_block_size>>>(
      device_element_variable_next.get_own(), device_element_volume.get_own(),
      thrust::raw_pointer_cast(device_element_variable_next_adapted.data()), thrust::raw_pointer_cast(device_element_volume_adapted.data()),
      device_element_adapt_data, num_new_elements);
  T8GPU_CUDA_CHECK_LAST_ERROR();
  T8GPU_CUDA_CHECK_ERROR(hipFree(device_element_adapt_data));
  device_element_variable_next = std::move(device_element_variable_next_adapted);
  device_element_volume = std::move(device_element_volume_adapted);

  forest = adapted_forest;

  device_element_variable_prev.resize(num_new_elements);

  device_element_fluxes.resize(num_new_elements);
  double* device_element_fluxes_ptr {device_element_fluxes.get_own()};
  T8GPU_CUDA_CHECK_ERROR(hipMemset(device_element_fluxes_ptr, 0, sizeof(double)*num_new_elements));

  device_element_refinement_criteria.resize(num_new_elements);

  num_ghost_elements = t8_forest_get_num_ghosts(forest);
  num_local_elements = t8_forest_get_local_num_elements(forest);
}

void t8gpu::AdvectionSolver::partition() {
  assert(t8_forest_is_committed(forest));
  t8_forest_ref(forest);
  t8_forest_t partitioned_forest {};
  t8_forest_init(&partitioned_forest);
  t8_forest_set_partition(partitioned_forest, forest, true);
  t8_forest_set_ghost(partitioned_forest, true, T8_GHOST_FACES);
  t8_forest_commit(partitioned_forest);

  t8_locidx_t num_old_elements {t8_forest_get_local_num_elements(forest)};
  t8_locidx_t num_new_elements {t8_forest_get_local_num_elements(partitioned_forest)};

  thrust::host_vector<t8_locidx_t> old_ranks(num_old_elements);
  thrust::host_vector<t8_locidx_t> old_indices(num_old_elements);
  for (t8_locidx_t i=0; i<num_old_elements; i++) {
    old_ranks[i] = rank;
    old_indices[i] = i;
  }

  thrust::host_vector<t8_locidx_t> new_ranks(num_new_elements);
  thrust::host_vector<t8_locidx_t> new_indices(num_new_elements);

  sc_array* sc_array_old_ranks_wrapper {sc_array_new_data(old_ranks.data(), sizeof(int), num_old_elements)};
  sc_array* sc_array_old_indices_wrapper {sc_array_new_data(old_indices.data(), sizeof(t8_locidx_t), num_old_elements)};

  sc_array* sc_array_new_ranks_wrapper {sc_array_new_data(new_ranks.data(), sizeof(int), num_new_elements)};
  sc_array* sc_array_new_indices_wrapper {sc_array_new_data(new_indices.data(), sizeof(t8_locidx_t), num_new_elements)};

  t8_forest_partition_data(forest, partitioned_forest,
			   sc_array_old_ranks_wrapper,
			   sc_array_new_ranks_wrapper);

  t8_forest_partition_data(forest, partitioned_forest,
			   sc_array_old_indices_wrapper,
			   sc_array_new_indices_wrapper);

  sc_array_destroy(sc_array_old_indices_wrapper);
  sc_array_destroy(sc_array_new_indices_wrapper);
  sc_array_destroy(sc_array_old_ranks_wrapper);
  sc_array_destroy(sc_array_new_ranks_wrapper);

  thrust::device_vector<int> device_new_ranks = new_ranks;
  thrust::device_vector<t8_locidx_t> device_new_indices = new_indices;

  t8gpu::SharedDeviceVector<double> device_new_element_variable(num_new_elements);
  t8gpu::SharedDeviceVector<double> device_new_element_volume(num_new_elements);

  constexpr int thread_block_size = 256;
  const int fluxes_num_blocks = (num_new_elements + thread_block_size - 1) / thread_block_size;
  partition_data<<<fluxes_num_blocks, thread_block_size>>>(thrust::raw_pointer_cast(device_new_ranks.data()),
							   thrust::raw_pointer_cast(device_new_indices.data()),
							   device_new_element_variable.get_own(),
							   device_new_element_volume.get_own(),
							   device_element_variable_next.get_all(),
							   device_element_volume.get_all(),
							   num_new_elements);
  hipDeviceSynchronize();
  MPI_Barrier(comm);

  device_element_variable_next = std::move(device_new_element_variable);
  device_element_variable_prev.resize(num_new_elements);
  device_element_volume = std::move(device_new_element_volume);
  device_element_refinement_criteria.resize(num_new_elements);
  device_element_fluxes.resize(num_new_elements);

  forest_user_data_t* forest_user_data = static_cast<forest_user_data_t*>(t8_forest_get_user_data(forest));
  t8_forest_set_user_data(partitioned_forest, forest_user_data);
  t8_forest_unref(&forest);
  forest = partitioned_forest;

  num_ghost_elements = t8_forest_get_num_ghosts(forest);
  num_local_elements = t8_forest_get_local_num_elements(forest);
}

void t8gpu::AdvectionSolver::compute_connectivity_information() {
  t8_locidx_t num_ghost_elements {t8_forest_get_num_ghosts(forest)};
  t8_locidx_t num_local_elements {t8_forest_get_local_num_elements(forest)};

  ranks.resize(num_local_elements + num_ghost_elements);
  indices.resize(num_local_elements + num_ghost_elements);
  for (t8_locidx_t i=0; i<num_local_elements; i++) {
    ranks[i] = rank;
    indices[i] = i;
  }
  sc_array* sc_array_ranks_wrapper {sc_array_new_data(ranks.data(), sizeof(int), num_local_elements + num_ghost_elements)};
  t8_forest_ghost_exchange_data(forest, sc_array_ranks_wrapper);
  sc_array_destroy(sc_array_ranks_wrapper);

  sc_array* sc_array_indices_wrapper {sc_array_new_data(indices.data(), sizeof(t8_locidx_t), num_local_elements + num_ghost_elements)};
  t8_forest_ghost_exchange_data(forest, sc_array_indices_wrapper);
  sc_array_destroy(sc_array_indices_wrapper);

  device_ranks = ranks;
  device_indices = indices;

  compute_edge_connectivity();
  device_face_neighbors = face_neighbors;
  device_face_normals = face_normals;
  device_face_area = face_area;
}

void t8gpu::AdvectionSolver::save_vtk(const std::string& prefix) const {
  thrust::host_vector<double> element_variable(num_local_elements);
  T8GPU_CUDA_CHECK_ERROR(hipMemcpy(element_variable.data(), device_element_variable_next.get_own(), sizeof(double)*element_variable.size(), hipMemcpyDeviceToHost));

  t8_vtk_data_field_t vtk_data_field {};
  vtk_data_field.type = T8_VTK_SCALAR;
  strcpy(vtk_data_field.description, "advection variable");
  vtk_data_field.data = element_variable.data();
  t8_forest_write_vtk_ext(forest, prefix.c_str(), 1, 1, 1, 1, 0, 0, 0, 1, &vtk_data_field);
}

double t8gpu::AdvectionSolver::compute_integral() const {
  double local_integral = 0.0;
  double const* mem {device_element_variable_next.get_own()};
  thrust::host_vector<double> variable(num_local_elements);
  T8GPU_CUDA_CHECK_ERROR(hipMemcpy(variable.data(), mem, sizeof(double)*device_element_variable_next.size(), hipMemcpyDeviceToHost));
  thrust::host_vector<double> volume(num_local_elements);
  T8GPU_CUDA_CHECK_ERROR(hipMemcpy(volume.data(), device_element_volume.get_own(), sizeof(double)*device_element_volume.size(), hipMemcpyDeviceToHost));

  for (t8_locidx_t i=0; i<num_local_elements; i++) {
    local_integral += volume[i] * variable[i];
  }
  double global_integral {};
  MPI_Allreduce(&local_integral, &global_integral, 1, MPI_DOUBLE, MPI_SUM, comm);
  return global_integral;
}

void t8gpu::AdvectionSolver::compute_edge_connectivity() {
  face_neighbors.clear();
  face_normals.clear();
  face_area.clear();

  assert(t8_forest_is_committed(forest));
  t8_locidx_t num_local_elements {t8_forest_get_local_num_elements(forest)};

  t8_locidx_t num_local_trees {t8_forest_get_num_local_trees(forest)};
  t8_locidx_t element_idx = 0;
  for (t8_locidx_t tree_idx = 0; tree_idx < num_local_trees; tree_idx++) {
    t8_eclass_t tree_class = t8_forest_get_tree_class(forest, tree_idx);
    t8_eclass_scheme_c* eclass_scheme {t8_forest_get_eclass_scheme(forest, tree_class)};

    t8_locidx_t num_elements_in_tree {t8_forest_get_tree_num_elements(forest, tree_idx)};
    for (t8_locidx_t tree_element_idx = 0; tree_element_idx < num_elements_in_tree; tree_element_idx++) {
      const t8_element_t* element {t8_forest_get_element_in_tree(forest, tree_idx, tree_element_idx)};

      t8_locidx_t num_faces {eclass_scheme->t8_element_num_faces(element)};
      for (t8_locidx_t face_idx = 0; face_idx < num_faces; face_idx++) {
        int num_neighbors {};
        int* dual_faces {};
        t8_locidx_t* neighbor_ids {};
        t8_element_t** neighbors {};
        t8_eclass_scheme_c* neigh_scheme {};

        t8_forest_leaf_face_neighbors(forest, tree_idx, element, &neighbors, face_idx, &dual_faces, &num_neighbors, &neighbor_ids, &neigh_scheme,
                                      true);

	for (int i=0; i<num_neighbors; i++) {
	  if (neighbor_ids[i] >= num_local_elements && rank < ranks[neighbor_ids[i]]) {
	    face_neighbors.push_back(element_idx);
	    face_neighbors.push_back(neighbor_ids[i]);
	    double face_normal[3];
	    t8_forest_element_face_normal(forest, tree_idx, element, face_idx, face_normal);
	    face_normals.push_back(face_normal[0]);
	    face_normals.push_back(face_normal[1]);
	    face_area.push_back(t8_forest_element_face_area(forest, tree_idx, element, face_idx) / static_cast<double>(num_neighbors));
	  }
	}

        if ((num_neighbors == 1) && (neighbor_ids[0] < num_local_elements) &&
            ((neighbor_ids[0] > element_idx) ||
             (neighbor_ids[0] < element_idx && neigh_scheme[0].t8_element_level(neighbors[0]) < eclass_scheme->t8_element_level(element)))) {
	  face_neighbors.push_back(element_idx);
	  face_neighbors.push_back(neighbor_ids[0]);
	  double face_normal[3];
	  t8_forest_element_face_normal(forest, tree_idx, element, face_idx, face_normal);
	  face_normals.push_back(face_normal[0]);
	  face_normals.push_back(face_normal[1]);
	  face_area.push_back(t8_forest_element_face_area(forest, tree_idx, element, face_idx));
        }
        T8_FREE(neighbors);
        T8_FREE(dual_faces);
        T8_FREE(neighbor_ids);
      }

      element_idx++;
    }
  }

  num_local_faces = face_area.size();
}

static int adapt_callback_initialization(t8_forest_t forest, t8_forest_t forest_from, t8_locidx_t which_tree, t8_locidx_t lelement_id,
					 t8_eclass_scheme_c* ts, const int is_family, const int num_elements, t8_element_t* elements[]) {
  t8_locidx_t element_level {ts->t8_element_level(elements[0])};

  double b = 0.02;

  if (element_level < t8gpu::AdvectionSolver::max_level) {
    double center[3];
    t8_forest_element_centroid(forest_from, which_tree, elements[0], center);

    double variable = sqrt((0.5 - center[0]) * (0.5 - center[0]) + (0.5 - center[1]) * (0.5 - center[1])) - 0.25;

    if (std::abs(variable) < b) return 1;
  }
  if (element_level > t8gpu::AdvectionSolver::min_level && is_family) {
    double center[] = {0.0, 0.0, 0.0};
    double current_element_center[] = {0.0, 0.0, 0.0};
    for (size_t i = 0; i < 4; i++) {
      t8_forest_element_centroid(forest_from, which_tree, elements[i], current_element_center);
      for (size_t j = 0; j < 3; j++) {
        center[j] += current_element_center[j] / 4.0;
      }
    }

    double variable = sqrt((0.5 - center[0]) * (0.5 - center[0]) + (0.5 - center[1]) * (0.5 - center[1])) - 0.25;

    if (std::abs(variable) > b) return -1;
  }

  return 0;
}

static int adapt_callback_iteration(t8_forest_t forest, t8_forest_t forest_from, t8_locidx_t which_tree, t8_locidx_t lelement_id, t8_eclass_scheme_c* ts,
				    const int is_family, const int num_elements, t8_element_t* elements[]) {
  forest_user_data_t* forest_user_data = static_cast<forest_user_data_t*>(t8_forest_get_user_data(forest_from));
  assert(forest_user_data != nullptr);

  t8_locidx_t element_level {ts->t8_element_level(elements[0])};

  t8_locidx_t tree_offset = t8_forest_get_tree_element_offset(forest_from, which_tree);

  double b = 1.0;
  double h = std::pow(0.5, element_level);

  if (element_level < t8gpu::AdvectionSolver::max_level) {
    double variable = (*forest_user_data->element_refinement_criteria)[tree_offset + lelement_id];

    if (std::abs(variable) < b * h) {
      return 1;
    }
  }
  if (element_level > t8gpu::AdvectionSolver::min_level && is_family) {
    double variable = 0.0;
    for (size_t i = 0; i < 4; i++) {
      variable += (*forest_user_data->element_refinement_criteria)[tree_offset + lelement_id + i] / 4.0;
    }

    if (std::abs(variable) > (2 * h) * b) {
      return -1;
    }
  }

  return 0;
}

__global__ static void compute_refinement_criteria(double const* __restrict__ variable, double* __restrict__ criteria, int nb_elements) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i >= nb_elements) return;

  criteria[i] = variable[i];
}

__global__ static void adapt_variable_and_volume(double const* __restrict__ variable_old, double const* __restrict__ volume_old,
                                                 double* __restrict__ variable_new, double* __restrict__ volume_new, t8_locidx_t* adapt_data,
                                                 int nb_new_elements) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i >= nb_new_elements) return;

  int diff = adapt_data[i + 1] - adapt_data[i];
  int nb_elements_sum = max(1, diff);

  volume_new[i] = volume_old[adapt_data[i]] * ((diff == 0 ? 0.25 : (diff == 1 ? 1.0 : 4.0)));
  if (i > 0 && adapt_data[i - 1] == adapt_data[i]) {
    volume_new[i] = volume_old[adapt_data[i]] * 0.25;
  }

  variable_new[i] = 0.0;
  for (int j = 0; j < nb_elements_sum; j++) {
    variable_new[i] += variable_old[adapt_data[i] + j] / static_cast<double>(nb_elements_sum);
  }
}

__global__ void partition_data(int* __restrict__ ranks, t8_locidx_t* __restrict__ indices,
			       double* __restrict__ new_variable, double* __restrict__ new_volume,
			       double const*const* __restrict__ old_variable, double const*const* __restrict__ old_volume,
			       int num_new_elements) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= num_new_elements) return;

  new_variable[i] = old_variable[ranks[i]][indices[i]];
  new_volume[i] = old_volume[ranks[i]][indices[i]];
}

__global__ static void compute_fluxes(double** __restrict__ variables, double** __restrict__ fluxes, double const* __restrict__ normal,
                                      double const* __restrict__ area, int const* e_idx, int* rank, t8_locidx_t* indices, int nb_edges) {
  double a[2] = {0.5 * sqrt(2.0), 0.5 * sqrt(2.0)};

  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= nb_edges) return;

  double flux = area[i] * (a[0] * normal[2 * i] + a[1] * normal[2 * i + 1]);

  if (flux > 0.0) {
    flux *= variables[rank[e_idx[2 * i]]][indices[e_idx[2 * i]]];
  } else {
    flux *= variables[rank[e_idx[2 * i + 1]]][indices[e_idx[2 * i + 1]]];
  }

  atomicAdd(&fluxes[rank[e_idx[2 * i]]][indices[e_idx[2 * i]]], -flux);
  atomicAdd(&fluxes[rank[e_idx[2 * i + 1]]][indices[e_idx[2 * i + 1]]], flux);
}

__global__ static void explicit_euler_time_step(double const* __restrict__ variable_prev, double* __restrict__ variable_next,
                                                double const* __restrict__ volume, double* __restrict__ fluxes, double delta_t, int nb_elements) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i >= nb_elements) return;

  variable_next[i] = variable_prev[i] + delta_t / volume[i] * fluxes[i];
}
