#include "hip/hip_runtime.h"
#include <advection_solver.h>
#include <t8.h>
#include <t8_cmesh.h>
#include <t8_cmesh/t8_cmesh_examples.h>
#include <t8_forest/t8_forest.h>
#include <t8_forest/t8_forest_iterate.h>
#include <t8_forest/t8_forest_partition.h>
#include <utils/hip/hip_runtime.h>
#include <utils/profiling.h>

#include <cassert>
#include <cmath>
#include <iostream>
#include <t8_schemes/t8_default/t8_default_cxx.hxx>
#include <timestepping/ssp_runge_kutta.h>

struct forest_user_data_t {
  thrust::host_vector<double>* element_refinement_criteria;
};

static int adapt_callback_initialization(t8_forest_t forest, t8_forest_t forest_from, t8_locidx_t which_tree, t8_locidx_t lelement_id,
					 t8_eclass_scheme_c* ts, const int is_family, const int num_elements, t8_element_t* elements[]);

static int adapt_callback_iteration(t8_forest_t forest, t8_forest_t forest_from, t8_locidx_t which_tree, t8_locidx_t lelement_id, t8_eclass_scheme_c* ts,
				    const int is_family, const int num_elements, t8_element_t* elements[]);

__global__ static void compute_refinement_criteria(double const* __restrict__ rho_v1,
						   double const* __restrict__ rho_v2,
						   double* __restrict__ criteria, int nb_elements);

__global__ static void adapt_variables_and_volume(double const* __restrict__ rho_old,
						  double const* __restrict__ rho_v1_old,
						  double const* __restrict__ rho_v2_old,
						  double const* __restrict__ rho_e_old,
						  double const* __restrict__ volume_old,
						  double* __restrict__ rho_new,
						  double* __restrict__ rho_v1_new,
						  double* __restrict__ rho_v2_new,
						  double* __restrict__ rho_e_new,
						  double* __restrict__ volume_new,
						  t8_locidx_t* adapt_data,
						  int nb_new_elements);

__global__ void partition_data(int* __restrict__ ranks, t8_locidx_t* __restrict__ indices,
			       double* __restrict__ new_rho,
			       double* __restrict__ new_rho_v1,
			       double* __restrict__ new_rho_v2,
			       double* __restrict__ new_rho_e,
			       double* __restrict__ new_volume,
			       double const*const* __restrict__ old_rho,
			       double const*const* __restrict__ old_rho_v1,
			       double const*const* __restrict__ old_rho_v2,
			       double const*const* __restrict__ old_rho_e,
			       double const*const* __restrict__ old_volume,
			       int num_new_elements);

__global__ static void hll_compute_fluxes(double** __restrict__ rho,
					  double** __restrict__ rho_v1,
					  double** __restrict__ rho_v2,
					  double** __restrict__ rho_e,
					  double** __restrict__ rho_fluxes,
					  double** __restrict__ rho_v1_fluxes,
					  double** __restrict__ rho_v2_fluxes,
					  double** __restrict__ rho_e_fluxes,
					  double const* __restrict__ normal,
					  double const* __restrict__ area,
					  int const* e_idx, int* rank,
					  t8_locidx_t* indices, int nb_edges);

__device__ static void kepes_compute_flux(double u_L[5],
					  double u_R[5],
					  double F_star[5],
					  double& uHat,
					  double& vHat,
					  double& wHat,
					  double& aHat,
					  double& rhoHat,
					  double& HHat,
					  double& p1Hat);

__global__ static void kepes_compute_fluxes(double** __restrict__ rho,
					    double** __restrict__ rho_v1,
					    double** __restrict__ rho_v2,
					    double** __restrict__ rho_e,
					    double** __restrict__ rho_fluxes,
					    double** __restrict__ rho_v1_fluxes,
					    double** __restrict__ rho_v2_fluxes,
					    double** __restrict__ rho_e_fluxes,
					    double const* __restrict__ normal,
					    double const* __restrict__ area,
					    int const* e_idx, int* rank,
					    t8_locidx_t* indices, int nb_edges);

t8gpu::AdvectionSolver::AdvectionSolver(sc_MPI_Comm comm)
    : comm(comm),
      cmesh(t8_cmesh_new_periodic(comm, 2)),
      scheme(t8_scheme_new_default_cxx()),
      forest(t8_forest_new_uniform(cmesh, scheme, 7, true, comm)),
      delta_t(0.2 * std::pow(0.5, max_level)) {
  t8_forest_t new_forest {};
  t8_forest_init(&new_forest);
  t8_forest_set_adapt(new_forest, forest, adapt_callback_initialization, true);
  t8_forest_set_ghost(new_forest, true, T8_GHOST_FACES);
  t8_forest_set_balance(new_forest, forest, false);
  t8_forest_set_partition(new_forest, forest, true);
  t8_forest_commit(new_forest);
  forest = new_forest;

  MPI_Comm_size(comm, &nb_ranks);
  MPI_Comm_rank(comm, &rank);

  rho_prev    = rho_0;
  rho_v1_prev = rho_v1_0;
  rho_v2_prev = rho_v2_0;
  rho_e_prev  = rho_e_0;

  rho_next    = rho_3;
  rho_v1_next = rho_v1_3;
  rho_v2_next = rho_v2_3;
  rho_e_next  = rho_e_3;

  num_ghost_elements = t8_forest_get_num_ghosts(forest);
  num_local_elements = t8_forest_get_local_num_elements(forest);

  ranks.resize(num_local_elements + num_ghost_elements);
  indices.resize(num_local_elements + num_ghost_elements);
  for (t8_locidx_t i=0; i<num_local_elements; i++) {
    ranks[i] = rank;
    indices[i] = i;
  }
  sc_array* sc_array_ranks_wrapper {sc_array_new_data(ranks.data(), sizeof(int), num_local_elements + num_ghost_elements)};
  t8_forest_ghost_exchange_data(forest, sc_array_ranks_wrapper);
  sc_array_destroy(sc_array_ranks_wrapper);

  sc_array* sc_array_indices_wrapper {sc_array_new_data(indices.data(), sizeof(t8_locidx_t), num_local_elements + num_ghost_elements)};
  t8_forest_ghost_exchange_data(forest, sc_array_indices_wrapper);
  sc_array_destroy(sc_array_indices_wrapper);

  device_ranks = ranks;
  device_indices = indices;

  thrust::host_vector<double> element_rho(num_local_elements);
  thrust::host_vector<double> element_rho_v1(num_local_elements);
  thrust::host_vector<double> element_rho_v2(num_local_elements);
  thrust::host_vector<double> element_rho_e(num_local_elements);

  thrust::host_vector<double> element_volume(num_local_elements);

  t8_locidx_t num_local_trees = t8_forest_get_num_local_trees(forest);
  t8_locidx_t element_idx = 0;
  for (t8_locidx_t tree_idx = 0; tree_idx < num_local_trees; tree_idx++) {
    t8_eclass_t tree_class {t8_forest_get_tree_class(forest, tree_idx)};
    t8_eclass_scheme_c* eclass_scheme {t8_forest_get_eclass_scheme(forest, tree_class)};

    t8_locidx_t num_elements_in_tree {t8_forest_get_tree_num_elements(forest, tree_idx)};
    for (t8_locidx_t tree_element_idx = 0; tree_element_idx < num_elements_in_tree; tree_element_idx++) {
      const t8_element_t* element {t8_forest_get_element_in_tree(forest, tree_idx, tree_element_idx)};

      double center[3];
      t8_forest_element_centroid(forest, tree_idx, element, center);

      double sigma = 0.05/sqrt(2.0);
      double gamma = 1.4;

      double x = center[0]-0.5;
      double y = center[1]-0.5;

      double rho = std::abs(y) < 0.25 ? 2.0 : 1.0;

      double v1 = std::abs(y) < 0.25 ? -0.5 : 0.5;
      double v2 = 0.1*sin(4.0*M_PI*x)*(exp(-((y-0.25)/(2*sigma))*((y-0.25)/(2*sigma)))+exp(-((y+0.25)/(2*sigma))*((y+0.25)/(2*sigma))));

      double rho_v1 = rho*v1;
      double rho_v2 = rho*v2;

      element_rho[element_idx]    = rho;
      element_rho_v1[element_idx] = rho_v1;
      element_rho_v2[element_idx] = rho_v2;
      element_rho_e[element_idx]  = 2.0/(gamma-1.0) + 0.5*(rho_v1 * rho_v1 + rho_v2 * rho_v2) / rho;


      element_volume[element_idx] = t8_forest_element_volume(forest, tree_idx, element);

      element_idx++;
    }
  }

  // resize shared and owned element variables
  device_element.resize(num_local_elements);

  element_refinement_criteria.resize(num_local_elements);
  device_element_refinement_criteria.resize(num_local_elements);

  // copy new shared element variables
  device_element.copy(rho_next, element_rho);
  device_element.copy(rho_v1_next, element_rho_v1);
  device_element.copy(rho_v2_next, element_rho_v2);
  device_element.copy(rho_e_next, element_rho_e);

  // device_element[volume] = element_volume;
  device_element.copy(volume, element_volume);

  // fill fluxes device element variable
  double* device_element_fluxes_ptr {device_element.get_own(rho_fluxes)};
  T8GPU_CUDA_CHECK_ERROR(hipMemset(device_element_fluxes_ptr, 0, 4*sizeof(double)*num_local_elements));

  compute_edge_connectivity();
  device_face_neighbors = face_neighbors;
  device_face_normals = face_normals;
  device_face_area = face_area;

  // TODO: remove allocation out of RAII paradigm
  forest_user_data_t* forest_user_data = static_cast<forest_user_data_t*>(malloc(sizeof(forest_user_data_t)));
  assert(forest_user_data != nullptr);

  forest_user_data->element_refinement_criteria = &element_refinement_criteria;
  t8_forest_set_user_data(forest, forest_user_data);
}

t8gpu::AdvectionSolver::~AdvectionSolver() {
  forest_user_data_t* forest_user_data {static_cast<forest_user_data_t*>(t8_forest_get_user_data(forest))};
  free(forest_user_data);

  t8_forest_unref(&forest);
  t8_cmesh_destroy(&cmesh);
}

void t8gpu::AdvectionSolver::iterate() {
  std::swap(rho_prev, rho_next);
  std::swap(rho_v1_prev, rho_v1_next);
  std::swap(rho_v2_prev, rho_v2_next);
  std::swap(rho_e_prev, rho_e_next);

  compute_fluxes(rho_prev,
		 rho_v1_prev,
		 rho_v2_prev,
		 rho_e_prev);

  constexpr int thread_block_size = 256;
  const int SSP_num_blocks = (num_local_elements + thread_block_size - 1) / thread_block_size;
  t8gpu::timestepping::SSP_3RK_step1<<<SSP_num_blocks, thread_block_size>>>(
      device_element.get_own(rho_prev),
      device_element.get_own(rho_v1_prev),
      device_element.get_own(rho_v2_prev),
      device_element.get_own(rho_e_prev),
      device_element.get_own(rho_1),
      device_element.get_own(rho_v1_1),
      device_element.get_own(rho_v2_1),
      device_element.get_own(rho_e_1),
      device_element.get_own(volume),
      device_element.get_own(rho_fluxes),
      device_element.get_own(rho_v1_fluxes),
      device_element.get_own(rho_v2_fluxes),
      device_element.get_own(rho_e_fluxes),
      delta_t, num_local_elements);
  T8GPU_CUDA_CHECK_LAST_ERROR();
  hipDeviceSynchronize();
  MPI_Barrier(comm);

  compute_fluxes(rho_1,
		 rho_v1_1,
		 rho_v2_1,
		 rho_e_1);

  t8gpu::timestepping::SSP_3RK_step2<<<SSP_num_blocks, thread_block_size>>>(
      device_element.get_own(rho_prev),
      device_element.get_own(rho_v1_prev),
      device_element.get_own(rho_v2_prev),
      device_element.get_own(rho_e_prev),
      device_element.get_own(rho_1),
      device_element.get_own(rho_v1_1),
      device_element.get_own(rho_v2_1),
      device_element.get_own(rho_e_1),
      device_element.get_own(rho_2),
      device_element.get_own(rho_v1_2),
      device_element.get_own(rho_v2_2),
      device_element.get_own(rho_e_2),
      device_element.get_own(volume),
      device_element.get_own(rho_fluxes),
      device_element.get_own(rho_v1_fluxes),
      device_element.get_own(rho_v2_fluxes),
      device_element.get_own(rho_e_fluxes),
      delta_t, num_local_elements);
  T8GPU_CUDA_CHECK_LAST_ERROR();
  hipDeviceSynchronize();
  MPI_Barrier(comm);

  compute_fluxes(rho_2,
		 rho_v1_2,
		 rho_v2_2,
		 rho_e_2);

  t8gpu::timestepping::SSP_3RK_step3<<<SSP_num_blocks, thread_block_size>>>(
      device_element.get_own(rho_prev),
      device_element.get_own(rho_v1_prev),
      device_element.get_own(rho_v2_prev),
      device_element.get_own(rho_e_prev),
      device_element.get_own(rho_2),
      device_element.get_own(rho_v1_2),
      device_element.get_own(rho_v2_2),
      device_element.get_own(rho_e_2),
      device_element.get_own(rho_next),
      device_element.get_own(rho_v1_next),
      device_element.get_own(rho_v2_next),
      device_element.get_own(rho_e_next),
      device_element.get_own(volume),
      device_element.get_own(rho_fluxes),
      device_element.get_own(rho_v1_fluxes),
      device_element.get_own(rho_v2_fluxes),
      device_element.get_own(rho_e_fluxes),
      delta_t, num_local_elements);
  T8GPU_CUDA_CHECK_LAST_ERROR();
}

__global__ void estimate_gradient(double const* const* __restrict__ rho,
				  double** __restrict__ rho_gradient,
				  double const* __restrict__ normal,
				  double const* __restrict__ area,
				  int const* e_idx, int* rank,
				  t8_locidx_t* indices, int nb_edges) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= nb_edges) return;

  int l_rank  = rank[e_idx[2 * i]];
  int l_index = indices[e_idx[2 * i]];

  int r_rank  = rank[e_idx[2 * i + 1]];
  int r_index = indices[e_idx[2 * i + 1]];

  double rho_l    = rho[l_rank][l_index];
  double rho_r    = rho[r_rank][r_index];

  double gradient = abs(rho_r - rho_l);

  atomicAdd(&rho_gradient[l_rank][l_index], gradient);
  atomicAdd(&rho_gradient[r_rank][r_index], gradient);
}

void t8gpu::AdvectionSolver::adapt() {
  constexpr int thread_block_size = 256;
  const int gradient_num_blocks = (num_local_faces + thread_block_size - 1) / thread_block_size;
  estimate_gradient<<<gradient_num_blocks, thread_block_size>>>(
	device_element.get_all(rho_next),
	device_element.get_all(rho_fluxes),
	thrust::raw_pointer_cast(device_face_normals.data()),
	thrust::raw_pointer_cast(device_face_area.data()),
	thrust::raw_pointer_cast(device_face_neighbors.data()),
	thrust::raw_pointer_cast(device_ranks.data()),
	thrust::raw_pointer_cast(device_indices.data()),
	num_local_faces);
  T8GPU_CUDA_CHECK_LAST_ERROR();
  hipDeviceSynchronize();
  MPI_Barrier(comm);

  const int fluxes_num_blocks = (num_local_elements + thread_block_size - 1) / thread_block_size;
  compute_refinement_criteria<<<fluxes_num_blocks, thread_block_size>>>(
	device_element.get_own(rho_fluxes),
	device_element.get_own(volume),
	thrust::raw_pointer_cast(device_element_refinement_criteria.data()),
	num_local_elements);
  T8GPU_CUDA_CHECK_LAST_ERROR();

  element_refinement_criteria = device_element_refinement_criteria;

  t8_forest_ref(forest);
  assert(t8_forest_is_committed(forest));

  t8_forest_t adapted_forest {};
  t8_forest_init(&adapted_forest);
  t8_forest_set_adapt(adapted_forest, forest, adapt_callback_iteration, false);
  t8_forest_set_ghost(adapted_forest, true, T8_GHOST_FACES);
  t8_forest_set_balance(adapted_forest, forest, true);
  t8_forest_commit(adapted_forest);

  t8_locidx_t old_idx = 0;
  t8_locidx_t new_idx = 0;

  t8_locidx_t num_new_elements {t8_forest_get_local_num_elements(adapted_forest)};
  t8_locidx_t num_old_elements {t8_forest_get_local_num_elements(forest)};

  thrust::host_vector<double> adapted_element_variable(num_new_elements);
  thrust::host_vector<double> adapted_element_volume(num_new_elements);
  thrust::host_vector<t8_locidx_t> element_adapt_data(num_new_elements + 1);

  thrust::host_vector<t8_locidx_t> old_levels(num_old_elements);
  thrust::host_vector<t8_locidx_t> new_levels(num_new_elements);

  t8_locidx_t num_old_local_trees = {t8_forest_get_num_local_trees(forest)};
  t8_locidx_t num_new_local_trees = {t8_forest_get_num_local_trees(forest)};

  t8_locidx_t current_idx = 0;
  for (t8_locidx_t tree_idx = 0; tree_idx < num_old_local_trees; tree_idx++) {
    t8_eclass_t old_tree_class {t8_forest_get_tree_class(forest, tree_idx)};
    t8_eclass_scheme_c* old_scheme = {t8_forest_get_eclass_scheme(forest, old_tree_class)};

    t8_locidx_t num_elements_in_tree {t8_forest_get_tree_num_elements(forest, tree_idx)};

    for (t8_locidx_t elem_idx = 0; elem_idx < num_elements_in_tree; elem_idx++) {
      t8_element_t const* element {t8_forest_get_element_in_tree(forest, tree_idx, elem_idx)};
      old_levels[current_idx] = old_scheme->t8_element_level(element);
      current_idx++;
    }
  }

  current_idx = 0;
  for (t8_locidx_t tree_idx = 0; tree_idx < num_new_local_trees; tree_idx++) {
    t8_eclass_t new_tree_class {t8_forest_get_tree_class(adapted_forest, tree_idx)};
    t8_eclass_scheme_c* new_scheme = {t8_forest_get_eclass_scheme(adapted_forest, new_tree_class)};

    t8_locidx_t num_elements_in_tree {t8_forest_get_tree_num_elements(adapted_forest, tree_idx)};

    for (t8_locidx_t elem_idx = 0; elem_idx < num_elements_in_tree; elem_idx++) {
      t8_element_t const* element {t8_forest_get_element_in_tree(adapted_forest, tree_idx, elem_idx)};
      new_levels[current_idx] = new_scheme->t8_element_level(element);
      current_idx++;
    }
  }

  while (old_idx < num_old_elements && new_idx < num_new_elements) {
    int old_level = old_levels[old_idx];
    int new_level = new_levels[new_idx];

    if (old_level < new_level) {  // refined
      for (size_t i = 0; i < 4; i++) {
        element_adapt_data[new_idx + i] = old_idx;
      }
      old_idx += 1;
      new_idx += 4;
    } else if (old_level > new_level) {  // coarsened
      for (size_t i = 0; i < 4; i++) {
      }
      element_adapt_data[new_idx] = old_idx;
      old_idx += 4;
      new_idx += 1;
    } else {
      element_adapt_data[new_idx] = old_idx;
      old_idx += 1;
      new_idx += 1;
    }
  }
  element_adapt_data[new_idx] = old_idx;

  forest_user_data_t* forest_user_data {static_cast<forest_user_data_t*>(t8_forest_get_user_data(forest))};
  assert(forest_user_data != nullptr);

  t8_forest_set_user_data(adapted_forest, forest_user_data);
  t8_forest_unref(&forest);

  thrust::device_vector<double> device_element_rho_next_adapted(num_new_elements);
  thrust::device_vector<double> device_element_rho_v1_next_adapted(num_new_elements);
  thrust::device_vector<double> device_element_rho_v2_next_adapted(num_new_elements);
  thrust::device_vector<double> device_element_rho_e_next_adapted(num_new_elements);

  thrust::device_vector<double> device_element_volume_adapted(num_new_elements);
  t8_locidx_t* device_element_adapt_data {};
  T8GPU_CUDA_CHECK_ERROR(hipMalloc(&device_element_adapt_data, (num_new_elements + 1) * sizeof(t8_locidx_t)));
  T8GPU_CUDA_CHECK_ERROR(
      hipMemcpy(device_element_adapt_data, element_adapt_data.data(), element_adapt_data.size() * sizeof(t8_locidx_t), hipMemcpyHostToDevice));
  const int adapt_num_blocks = (num_new_elements + thread_block_size - 1) / thread_block_size;
  adapt_variables_and_volume<<<adapt_num_blocks, thread_block_size>>>(
      device_element.get_own(rho_next),
      device_element.get_own(rho_v1_next),
      device_element.get_own(rho_v2_next),
      device_element.get_own(rho_e_next),
      device_element.get_own(volume),
      thrust::raw_pointer_cast(device_element_rho_next_adapted.data()),
      thrust::raw_pointer_cast(device_element_rho_v1_next_adapted.data()),
      thrust::raw_pointer_cast(device_element_rho_v2_next_adapted.data()),
      thrust::raw_pointer_cast(device_element_rho_e_next_adapted.data()),
      thrust::raw_pointer_cast(device_element_volume_adapted.data()),
      device_element_adapt_data, num_new_elements);
  T8GPU_CUDA_CHECK_LAST_ERROR();
  T8GPU_CUDA_CHECK_ERROR(hipFree(device_element_adapt_data));

  // resize shared and owned element variables
  device_element.resize(num_new_elements);

  element_refinement_criteria.resize(num_new_elements);
  device_element_refinement_criteria.resize(num_new_elements);

  // fill fluxes device element variable
  double* device_element_rho_fluxes_ptr {device_element.get_own(rho_fluxes)};
  T8GPU_CUDA_CHECK_ERROR(hipMemset(device_element_rho_fluxes_ptr, 0, 4*sizeof(double)*num_new_elements));

  // TODO add copy with rvalue reference
  device_element.copy(rho_next, std::move(device_element_rho_next_adapted));
  device_element.copy(rho_v1_next, std::move(device_element_rho_v1_next_adapted));
  device_element.copy(rho_v2_next, std::move(device_element_rho_v2_next_adapted));
  device_element.copy(rho_e_next, std::move(device_element_rho_e_next_adapted));
  device_element.copy(volume, std::move(device_element_volume_adapted));


  forest = adapted_forest;

  num_ghost_elements = t8_forest_get_num_ghosts(forest);
  num_local_elements = t8_forest_get_local_num_elements(forest);
}

void t8gpu::AdvectionSolver::partition() {
  assert(t8_forest_is_committed(forest));
  t8_forest_ref(forest);
  t8_forest_t partitioned_forest {};
  t8_forest_init(&partitioned_forest);
  t8_forest_set_partition(partitioned_forest, forest, true);
  t8_forest_set_ghost(partitioned_forest, true, T8_GHOST_FACES);
  t8_forest_commit(partitioned_forest);

  t8_locidx_t num_old_elements {t8_forest_get_local_num_elements(forest)};
  t8_locidx_t num_new_elements {t8_forest_get_local_num_elements(partitioned_forest)};

  thrust::host_vector<t8_locidx_t> old_ranks(num_old_elements);
  thrust::host_vector<t8_locidx_t> old_indices(num_old_elements);
  for (t8_locidx_t i=0; i<num_old_elements; i++) {
    old_ranks[i] = rank;
    old_indices[i] = i;
  }

  thrust::host_vector<t8_locidx_t> new_ranks(num_new_elements);
  thrust::host_vector<t8_locidx_t> new_indices(num_new_elements);

  sc_array* sc_array_old_ranks_wrapper {sc_array_new_data(old_ranks.data(), sizeof(int), num_old_elements)};
  sc_array* sc_array_old_indices_wrapper {sc_array_new_data(old_indices.data(), sizeof(t8_locidx_t), num_old_elements)};

  sc_array* sc_array_new_ranks_wrapper {sc_array_new_data(new_ranks.data(), sizeof(int), num_new_elements)};
  sc_array* sc_array_new_indices_wrapper {sc_array_new_data(new_indices.data(), sizeof(t8_locidx_t), num_new_elements)};

  t8_forest_partition_data(forest, partitioned_forest,
			   sc_array_old_ranks_wrapper,
			   sc_array_new_ranks_wrapper);

  t8_forest_partition_data(forest, partitioned_forest,
			   sc_array_old_indices_wrapper,
			   sc_array_new_indices_wrapper);

  sc_array_destroy(sc_array_old_indices_wrapper);
  sc_array_destroy(sc_array_new_indices_wrapper);
  sc_array_destroy(sc_array_old_ranks_wrapper);
  sc_array_destroy(sc_array_new_ranks_wrapper);

  thrust::device_vector<int> device_new_ranks = new_ranks;
  thrust::device_vector<t8_locidx_t> device_new_indices = new_indices;

  thrust::device_vector<double> device_new_element_rho(num_new_elements);
  thrust::device_vector<double> device_new_element_rho_v1(num_new_elements);
  thrust::device_vector<double> device_new_element_rho_v2(num_new_elements);
  thrust::device_vector<double> device_new_element_rho_e(num_new_elements);
  thrust::device_vector<double> device_new_element_volume(num_new_elements);

  constexpr int thread_block_size = 256;
  const int fluxes_num_blocks = (num_new_elements + thread_block_size - 1) / thread_block_size;
  partition_data<<<fluxes_num_blocks, thread_block_size>>>(thrust::raw_pointer_cast(device_new_ranks.data()),
							   thrust::raw_pointer_cast(device_new_indices.data()),
							   thrust::raw_pointer_cast(device_new_element_rho.data()),
							   thrust::raw_pointer_cast(device_new_element_rho_v1.data()),
							   thrust::raw_pointer_cast(device_new_element_rho_v2.data()),
							   thrust::raw_pointer_cast(device_new_element_rho_e.data()),
							   thrust::raw_pointer_cast(device_new_element_volume.data()),
							   device_element.get_all(rho_next),
							   device_element.get_all(rho_v1_next),
							   device_element.get_all(rho_v2_next),
							   device_element.get_all(rho_e_next),
							   device_element.get_all(volume),
							   num_new_elements);
  hipDeviceSynchronize();
  MPI_Barrier(comm);

  // resize shared and own element variables
  device_element.resize(num_new_elements);

  device_element_refinement_criteria.resize(num_new_elements);

  // copy new shared element variables
  device_element.copy(rho_next, std::move(device_new_element_rho));
  device_element.copy(rho_v1_next, std::move(device_new_element_rho_v1));
  device_element.copy(rho_v2_next, std::move(device_new_element_rho_v2));
  device_element.copy(rho_e_next, std::move(device_new_element_rho_e));
  device_element.copy(volume, std::move(device_new_element_volume));

  double* device_element_rho_fluxes_ptr {device_element.get_own(rho_fluxes)};
  T8GPU_CUDA_CHECK_ERROR(hipMemset(device_element_rho_fluxes_ptr, 0, 4*sizeof(double)*num_new_elements));

  forest_user_data_t* forest_user_data = static_cast<forest_user_data_t*>(t8_forest_get_user_data(forest));
  t8_forest_set_user_data(partitioned_forest, forest_user_data);
  t8_forest_unref(&forest);
  forest = partitioned_forest;

  num_ghost_elements = t8_forest_get_num_ghosts(forest);
  num_local_elements = t8_forest_get_local_num_elements(forest);
}

void t8gpu::AdvectionSolver::compute_connectivity_information() {
  t8_locidx_t num_ghost_elements {t8_forest_get_num_ghosts(forest)};
  t8_locidx_t num_local_elements {t8_forest_get_local_num_elements(forest)};

  ranks.resize(num_local_elements + num_ghost_elements);
  indices.resize(num_local_elements + num_ghost_elements);
  for (t8_locidx_t i=0; i<num_local_elements; i++) {
    ranks[i] = rank;
    indices[i] = i;
  }
  sc_array* sc_array_ranks_wrapper {sc_array_new_data(ranks.data(), sizeof(int), num_local_elements + num_ghost_elements)};
  t8_forest_ghost_exchange_data(forest, sc_array_ranks_wrapper);
  sc_array_destroy(sc_array_ranks_wrapper);

  sc_array* sc_array_indices_wrapper {sc_array_new_data(indices.data(), sizeof(t8_locidx_t), num_local_elements + num_ghost_elements)};
  t8_forest_ghost_exchange_data(forest, sc_array_indices_wrapper);
  sc_array_destroy(sc_array_indices_wrapper);

  device_ranks = ranks;
  device_indices = indices;

  compute_edge_connectivity();
  device_face_neighbors = face_neighbors;
  device_face_normals = face_normals;
  device_face_area = face_area;
}

void t8gpu::AdvectionSolver::save_vtk(const std::string& prefix) const {
  thrust::host_vector<double> element_variable(num_local_elements);
  T8GPU_CUDA_CHECK_ERROR(hipMemcpy(element_variable.data(), device_element.get_own(rho_next), sizeof(double)*element_variable.size(), hipMemcpyDeviceToHost));

  t8_vtk_data_field_t vtk_data_field {};
  vtk_data_field.type = T8_VTK_SCALAR;
  strcpy(vtk_data_field.description, "advection variable");
  vtk_data_field.data = element_variable.data();
  t8_forest_write_vtk_ext(forest, prefix.c_str(), 1, 1, 1, 1, 0, 0, 0, 1, &vtk_data_field);
}

double t8gpu::AdvectionSolver::compute_integral() const {
  double local_integral = 0.0;
  double const* mem {device_element.get_own(rho_next)};
  thrust::host_vector<double> variable(num_local_elements);
  T8GPU_CUDA_CHECK_ERROR(hipMemcpy(variable.data(), mem, sizeof(double)*device_element.size(), hipMemcpyDeviceToHost));
  thrust::host_vector<double> volume(num_local_elements);
  T8GPU_CUDA_CHECK_ERROR(hipMemcpy(volume.data(), device_element.get_own(VariableName::volume), sizeof(double)*device_element.size(), hipMemcpyDeviceToHost));

  for (t8_locidx_t i=0; i<num_local_elements; i++) {
    local_integral += volume[i] * variable[i];
  }
  double global_integral {};
  MPI_Allreduce(&local_integral, &global_integral, 1, MPI_DOUBLE, MPI_SUM, comm);
  return global_integral;
}

void t8gpu::AdvectionSolver::compute_edge_connectivity() {
  face_neighbors.clear();
  face_normals.clear();
  face_area.clear();

  assert(t8_forest_is_committed(forest));
  t8_locidx_t num_local_elements {t8_forest_get_local_num_elements(forest)};

  t8_locidx_t num_local_trees {t8_forest_get_num_local_trees(forest)};
  t8_locidx_t element_idx = 0;
  for (t8_locidx_t tree_idx = 0; tree_idx < num_local_trees; tree_idx++) {
    t8_eclass_t tree_class = t8_forest_get_tree_class(forest, tree_idx);
    t8_eclass_scheme_c* eclass_scheme {t8_forest_get_eclass_scheme(forest, tree_class)};

    t8_locidx_t num_elements_in_tree {t8_forest_get_tree_num_elements(forest, tree_idx)};
    for (t8_locidx_t tree_element_idx = 0; tree_element_idx < num_elements_in_tree; tree_element_idx++) {
      const t8_element_t* element {t8_forest_get_element_in_tree(forest, tree_idx, tree_element_idx)};

      t8_locidx_t num_faces {eclass_scheme->t8_element_num_faces(element)};
      for (t8_locidx_t face_idx = 0; face_idx < num_faces; face_idx++) {
        int num_neighbors {};
        int* dual_faces {};
        t8_locidx_t* neighbor_ids {};
        t8_element_t** neighbors {};
        t8_eclass_scheme_c* neigh_scheme {};

        t8_forest_leaf_face_neighbors(forest, tree_idx, element, &neighbors, face_idx, &dual_faces, &num_neighbors, &neighbor_ids, &neigh_scheme,
                                      true);

	for (int i=0; i<num_neighbors; i++) {
	  if (neighbor_ids[i] >= num_local_elements && rank < ranks[neighbor_ids[i]]) {
	    face_neighbors.push_back(element_idx);
	    face_neighbors.push_back(neighbor_ids[i]);
	    double face_normal[3];
	    t8_forest_element_face_normal(forest, tree_idx, element, face_idx, face_normal);
	    face_normals.push_back(face_normal[0]);
	    face_normals.push_back(face_normal[1]);
	    face_area.push_back(t8_forest_element_face_area(forest, tree_idx, element, face_idx) / static_cast<double>(num_neighbors));
	  }
	}

        if ((num_neighbors == 1) && (neighbor_ids[0] < num_local_elements) &&
            ((neighbor_ids[0] > element_idx) ||
             (neighbor_ids[0] < element_idx && neigh_scheme[0].t8_element_level(neighbors[0]) < eclass_scheme->t8_element_level(element)))) {
	  face_neighbors.push_back(element_idx);
	  face_neighbors.push_back(neighbor_ids[0]);
	  double face_normal[3];
	  t8_forest_element_face_normal(forest, tree_idx, element, face_idx, face_normal);
	  face_normals.push_back(face_normal[0]);
	  face_normals.push_back(face_normal[1]);
	  face_area.push_back(t8_forest_element_face_area(forest, tree_idx, element, face_idx));
        }
        T8_FREE(neighbors);
        T8_FREE(dual_faces);
        T8_FREE(neighbor_ids);
      }

      element_idx++;
    }
  }

  num_local_faces = face_area.size();
}

void t8gpu::AdvectionSolver::compute_fluxes(VariableName rho,
					    VariableName rho_v1,
					    VariableName rho_v2,
					    VariableName rho_e) {
  constexpr int thread_block_size = 256;
  const int fluxes_num_blocks = (num_local_faces + thread_block_size - 1) / thread_block_size;
  kepes_compute_fluxes<<<fluxes_num_blocks, thread_block_size>>>(
								 device_element.get_all(rho),
								 device_element.get_all(rho_v1),
								 device_element.get_all(rho_v2),
								 device_element.get_all(rho_e),
								 device_element.get_all(rho_fluxes),
								 device_element.get_all(rho_v1_fluxes),
								 device_element.get_all(rho_v2_fluxes),
								 device_element.get_all(rho_e_fluxes),
								 thrust::raw_pointer_cast(device_face_normals.data()),
								 thrust::raw_pointer_cast(device_face_area.data()),
								 thrust::raw_pointer_cast(device_face_neighbors.data()),
								 thrust::raw_pointer_cast(device_ranks.data()),
								 thrust::raw_pointer_cast(device_indices.data()),
								 num_local_faces);
  T8GPU_CUDA_CHECK_LAST_ERROR();
  hipDeviceSynchronize();
  MPI_Barrier(comm);
}

static int adapt_callback_initialization(t8_forest_t forest, t8_forest_t forest_from, t8_locidx_t which_tree, t8_locidx_t lelement_id,
					 t8_eclass_scheme_c* ts, const int is_family, const int num_elements, t8_element_t* elements[]) {
  // t8_locidx_t element_level {ts->t8_element_level(elements[0])};

  // double b = 0.02;

  // if (element_level < t8gpu::AdvectionSolver::max_level) {
  //   double center[3];
  //   t8_forest_element_centroid(forest_from, which_tree, elements[0], center);

  //   double variable = sqrt((0.5 - center[0]) * (0.5 - center[0]) + (0.5 - center[1]) * (0.5 - center[1])) - 0.25;

  //   if (std::abs(variable) < b) return 1;
  // }
  // if (element_level > t8gpu::AdvectionSolver::min_level && is_family) {
  //   double center[] = {0.0, 0.0, 0.0};
  //   double current_element_center[] = {0.0, 0.0, 0.0};
  //   for (size_t i = 0; i < 4; i++) {
  //     t8_forest_element_centroid(forest_from, which_tree, elements[i], current_element_center);
  //     for (size_t j = 0; j < 3; j++) {
  //       center[j] += current_element_center[j] / 4.0;
  //     }
  //   }

  //   double variable = sqrt((0.5 - center[0]) * (0.5 - center[0]) + (0.5 - center[1]) * (0.5 - center[1])) - 0.25;

  //   if (std::abs(variable) > b) return -1;
  // }

  return 0;
}

static int adapt_callback_iteration(t8_forest_t forest, t8_forest_t forest_from, t8_locidx_t which_tree, t8_locidx_t lelement_id, t8_eclass_scheme_c* ts,
				    const int is_family, const int num_elements, t8_element_t* elements[]) {
  forest_user_data_t* forest_user_data = static_cast<forest_user_data_t*>(t8_forest_get_user_data(forest_from));
  assert(forest_user_data != nullptr);

  t8_locidx_t element_level {ts->t8_element_level(elements[0])};

  t8_locidx_t tree_offset = t8_forest_get_tree_element_offset(forest_from, which_tree);

  double b = 10.0;

  if (element_level < t8gpu::AdvectionSolver::max_level) {
    double criteria = (*forest_user_data->element_refinement_criteria)[tree_offset + lelement_id];

    if (criteria > b) {
      return 1;
    }
  }
  if (element_level > t8gpu::AdvectionSolver::min_level && is_family) {
    double criteria = 0.0;
    for (size_t i = 0; i < 4; i++) {
      criteria += (*forest_user_data->element_refinement_criteria)[tree_offset + lelement_id + i] / 4.0;
    }

    if (criteria < b) {
      return -1;
    }
  }

  return 0;
}

__global__ static void compute_refinement_criteria(double const* __restrict__ fluxes_rho,
						   double const* __restrict__ volume,
						   double* __restrict__ criteria, int nb_elements) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i >= nb_elements) return;

  criteria[i] = fluxes_rho[i] / sqrt(volume[i]);
}

__global__ static void adapt_variables_and_volume(double const* __restrict__ rho_old,
						  double const* __restrict__ rho_v1_old,
						  double const* __restrict__ rho_v2_old,
						  double const* __restrict__ rho_e_old,
						  double const* __restrict__ volume_old,
						  double* __restrict__ rho_new,
						  double* __restrict__ rho_v1_new,
						  double* __restrict__ rho_v2_new,
						  double* __restrict__ rho_e_new,
						  double* __restrict__ volume_new,
						  t8_locidx_t* adapt_data,
						  int nb_new_elements) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i >= nb_new_elements) return;

  int diff = adapt_data[i + 1] - adapt_data[i];
  int nb_elements_sum = max(1, diff);

  volume_new[i] = volume_old[adapt_data[i]] * ((diff == 0 ? 0.25 : (diff == 1 ? 1.0 : 4.0)));
  if (i > 0 && adapt_data[i - 1] == adapt_data[i]) {
    volume_new[i] = volume_old[adapt_data[i]] * 0.25;
  }

  rho_new[i] = 0.0;
  rho_v1_new[i] = 0.0;
  rho_v2_new[i] = 0.0;
  rho_e_new[i] = 0.0;
  for (int j = 0; j < nb_elements_sum; j++) {
    rho_new[i]    += rho_old[adapt_data[i] + j] / static_cast<double>(nb_elements_sum);
    rho_v1_new[i] += rho_v1_old[adapt_data[i] + j] / static_cast<double>(nb_elements_sum);
    rho_v2_new[i] += rho_v2_old[adapt_data[i] + j] / static_cast<double>(nb_elements_sum);
    rho_e_new[i]  += rho_e_old[adapt_data[i] + j] / static_cast<double>(nb_elements_sum);
  }
}

__global__ void partition_data(int* __restrict__ ranks, t8_locidx_t* __restrict__ indices,
			       double* __restrict__ new_rho,
			       double* __restrict__ new_rho_v1,
			       double* __restrict__ new_rho_v2,
			       double* __restrict__ new_rho_e,
			       double* __restrict__ new_volume,
			       double const*const* __restrict__ old_rho,
			       double const*const* __restrict__ old_rho_v1,
			       double const*const* __restrict__ old_rho_v2,
			       double const*const* __restrict__ old_rho_e,
			       double const*const* __restrict__ old_volume,
			       int num_new_elements) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= num_new_elements) return;

  new_rho[i]    = old_rho[ranks[i]][indices[i]];
  new_rho_v1[i] = old_rho_v1[ranks[i]][indices[i]];
  new_rho_v2[i] = old_rho_v2[ranks[i]][indices[i]];
  new_rho_e[i]  = old_rho_e[ranks[i]][indices[i]];

  new_volume[i] = old_volume[ranks[i]][indices[i]];
}

__global__ static void hll_compute_fluxes(double** __restrict__ rho,
					  double** __restrict__ rho_v1,
					  double** __restrict__ rho_v2,
					  double** __restrict__ rho_e,
					  double** __restrict__ rho_fluxes,
					  double** __restrict__ rho_v1_fluxes,
					  double** __restrict__ rho_v2_fluxes,
					  double** __restrict__ rho_e_fluxes,
					  double const* __restrict__ normal,
					  double const* __restrict__ area,
					  int const* e_idx, int* rank,
					  t8_locidx_t* indices, int nb_edges) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= nb_edges) return;

  double gamma = 1.4;

  double face_surface = area[i];

  int l_rank  = rank[e_idx[2 * i]];
  int l_index = indices[e_idx[2 * i]];

  int r_rank  = rank[e_idx[2 * i + 1]];
  int r_index = indices[e_idx[2 * i + 1]];

  double nx = normal[2*i];
  double ny = normal[2*i+1];

  double rho_l    = rho[l_rank][l_index];
  double rho_vx_l = rho_v1[l_rank][l_index];
  double rho_vy_l = rho_v2[l_rank][l_index];
  double rho_e_l  = rho_e[l_rank][l_index];

  double rho_r    = rho[r_rank][r_index];
  double rho_vx_r = rho_v1[r_rank][r_index];
  double rho_vy_r = rho_v2[r_rank][r_index];
  double rho_e_r  = rho_e[r_rank][r_index];

  // rotate from (x,y) basis to local basis (n,t)
  double rho_v1_l =  nx*rho_vx_l + ny*rho_vy_l;
  double rho_v2_l = -ny*rho_vx_l + nx*rho_vy_l;

  double rho_v1_r =  nx*rho_vx_r + ny*rho_vy_r;
  double rho_v2_r = -ny*rho_vx_r + nx*rho_vy_r;

  double v1_l = rho_v1_l/rho_l;
  double v2_l = rho_v2_l/rho_l;
  double p_l = (gamma-1)*(rho_e_l-0.5*rho_l*(v1_l*v1_l+v2_l*v2_l));
  double H_l = (rho_e_l+p_l)/rho_l;
  double c_l = sqrt((gamma-1)*(H_l-0.5*(v1_l*v1_l+v2_l*v2_l)));

  double v1_r = rho_v1_r/rho_r;
  double v2_r = rho_v2_r/rho_r;
  double p_r = (gamma-1)*(rho_e_r-0.5*rho_r*(v1_r*v1_r+v2_r*v2_r));
  double H_r = (rho_e_r+p_r)/rho_r;
  double c_r = sqrt((gamma-1)*(H_r-0.5*(v1_r*v1_r+v2_r*v2_r)));

  double sqrt_rho_l = sqrt(rho_l);
  double sqrt_rho_r = sqrt(rho_r);
  double sum_weights = sqrt_rho_l+sqrt_rho_r;

  double v1_roe = (sqrt_rho_l*v1_l+sqrt_rho_r*v1_r)/sum_weights;
  double v2_roe = (sqrt_rho_l*v2_l+sqrt_rho_r*v2_r)/sum_weights;
  double H_roe = (sqrt_rho_l*H_l+sqrt_rho_r*H_r)/sum_weights;
  double c_roe = sqrt((gamma-1)*(H_roe-0.5*(v1_roe*v1_roe+v2_roe*v2_roe)));

  double S_l = min(v1_roe-c_roe, v1_l-c_l);
  double S_r = max(v1_roe+c_roe, v1_r+c_r);

  // double x_wave_speeds[i,j-1] = max(-S_l, S_r);

  double F_l[4] = {rho_v1_l,
    rho_v1_l*rho_v1_l/rho_l + p_l,
    rho_v1_l*v2_l,
    rho_v1_l*H_l};

  double F_r[4] = {rho_v1_r,
    rho_v1_r*rho_v1_r/rho_r + p_r,
    rho_v1_r*v2_r,
    rho_v1_r*H_r};

  double S_l_clamp = min(S_l, 0.0);
  double S_r_clamp = max(S_r, 0.0);

  double rho_flux    = face_surface*((S_r_clamp*F_l[0]-S_l_clamp*F_r[0])+S_r_clamp*S_l_clamp*(rho_r-rho_l))/(S_r_clamp-S_l_clamp);
  double rho_v1_flux = face_surface*((S_r_clamp*F_l[1]-S_l_clamp*F_r[1])+S_r_clamp*S_l_clamp*(rho_v1_r-rho_v1_l))/(S_r_clamp-S_l_clamp);
  double rho_v2_flux = face_surface*((S_r_clamp*F_l[2]-S_l_clamp*F_r[2])+S_r_clamp*S_l_clamp*(rho_v2_r-rho_v2_l))/(S_r_clamp-S_l_clamp);
  double rho_e_flux  = face_surface*((S_r_clamp*F_l[3]-S_l_clamp*F_r[3])+S_r_clamp*S_l_clamp*(rho_e_r-rho_e_l))/(S_r_clamp-S_l_clamp);

  // rotate back
  double rho_vx_flux = nx*rho_v1_flux - ny*rho_v2_flux;
  double rho_vy_flux = ny*rho_v1_flux + nx*rho_v2_flux;

  atomicAdd(&rho_fluxes[l_rank][l_index], -rho_flux);
  atomicAdd(&rho_fluxes[r_rank][r_index],  rho_flux);

  atomicAdd(&rho_v1_fluxes[l_rank][l_index], -rho_vx_flux);
  atomicAdd(&rho_v1_fluxes[r_rank][r_index],  rho_vx_flux);

  atomicAdd(&rho_v2_fluxes[l_rank][l_index], -rho_vy_flux);
  atomicAdd(&rho_v2_fluxes[r_rank][r_index],  rho_vy_flux);

  atomicAdd(&rho_e_fluxes[l_rank][l_index], -rho_e_flux);
  atomicAdd(&rho_e_fluxes[r_rank][r_index],  rho_e_flux);
}

__device__ static double ln_mean(double aL, double aR) {
  double Xi = aR/aL;
  double u = (Xi*(Xi-2.0)+1.0)/(Xi*(Xi+2.0)+1.0);

  double eps = 1.0e-4;
  if (u < eps) {
    return (aL+aR)*52.50/(105.0 + u*(35.0 + u*(21.0 + u*15.0)));
  } else {
    return (aR-aL)/log(Xi);
  }
}

__device__ static void kepes_compute_flux(double u_L[5],
					  double u_R[5],
					  double F_star[5],
					  double& uHat,
					  double& vHat,
					  double& wHat,
					  double& aHat,
					  double& rhoHat,
					  double& HHat,
					  double& p1Hat) {
  double kappa = 1.4;
  double kappaM1 = kappa - 1.0;
  double sKappaM1 = 1.0/kappaM1;

  double sRho_L = 1.0/u_L[0];
  double velU_L = sRho_L*u_L[1];
  double velV_L = sRho_L*u_L[2];
  double velW_L = sRho_L*u_L[3];

  double sRho_R = 1.0/u_R[0];
  double velU_R = sRho_R*u_R[1];
  double velV_R = sRho_R*u_R[2];
  double velW_R = sRho_R*u_R[3];

  double Vel2s2_L = 0.5*(velU_L*velU_L+velV_L*velV_L+velW_L*velW_L);
  double Vel2s2_R = 0.5*(velU_R*velU_R+velV_R*velV_R+velW_R*velW_R);

  double p_L = kappaM1*(u_L[4] - u_L[0]*Vel2s2_L);
  double p_R = kappaM1*(u_R[4] - u_R[0]*Vel2s2_R);

  double beta_L = 0.5*u_L[0]/p_L;
  double beta_R = 0.5*u_R[0]/p_R;

  double rho_MEAN  = 0.5*(u_L[0]+u_R[0]);
  rhoHat    = ln_mean(u_L[0],u_R[0]);
  double beta_MEAN = 0.5*(beta_L+beta_R);
  double beta_Hat  = ln_mean(beta_L,beta_R);

  uHat      = 0.5*(velU_L+velU_R);
  vHat      = 0.5*(velV_L+velV_R);
  wHat      = 0.5*(velW_L+velW_R);
  aHat      = sqrt(kappa*0.5*(p_L+p_R)/rhoHat);
  HHat      = kappa/(2.0*kappaM1*beta_Hat) + 0.5*(velU_L*velU_R+velV_L*velV_R+velW_L*velW_R);
  p1Hat     = 0.5*rho_MEAN/beta_MEAN;
  double Vel2_M    = Vel2s2_L+Vel2s2_R;

  double qHat      = uHat;
  F_star[0] = rhoHat*qHat;
  F_star[1] = F_star[0]*uHat + p1Hat;
  F_star[2] = F_star[0]*vHat;
  F_star[3] = F_star[0]*wHat;
  F_star[4] = F_star[0]*0.5*(sKappaM1/beta_Hat - Vel2_M) + uHat*F_star[1] + vHat*F_star[2] + wHat*F_star[3];
}

__device__ static void kepes_compute_diffusion_matrix(double u_L[5],
						      double u_R[5],
						      double F_star[5],
						      double RHat[5][5],
						      double DHat[5]) {

  double uHat;
  double vHat;
  double wHat;
  double aHat;
  double rhoHat;
  double hHat;
  double p1Hat;

  double kappa = 1.4;
  double kappaM1 = kappa - 1.0;

  kepes_compute_flux(u_L,
		     u_R,
		     F_star,
		     uHat,
		     vHat,
		     wHat,
		     aHat,
		     rhoHat,
		     hHat,
		     p1Hat);

  double R_hat[5][5] = {
    {           1.0,                                     1.0,  0.0,  0.0,            1.0},
    {     uHat-aHat,                                    uHat,  0.0,  0.0,      uHat+aHat},
    {          vHat,                                    vHat,  1.0,  0.0,           vHat},
    {          wHat,                                    wHat,  0.0,  1.0,           wHat},
    {hHat-uHat*aHat, 0.5*(uHat*uHat + vHat*vHat + wHat*wHat), vHat, wHat, hHat+uHat*aHat}
  };

  for (size_t i=0; i<5; i++)
    for (size_t j=0; j<5; j++)
      RHat[i][j] = R_hat[i][j];

  DHat[0] = 0.5*abs(uHat-aHat)*rhoHat/kappa;
  DHat[1] = abs(uHat)*(kappaM1/kappa)*rhoHat;
  DHat[2] = abs(uHat)*p1Hat;
  DHat[3] = DHat[2];
  DHat[4] = 0.5*abs(uHat+aHat)*rhoHat/kappa;

}

__global__ static void kepes_compute_fluxes(double** __restrict__ rho,
					    double** __restrict__ rho_v1,
					    double** __restrict__ rho_v2,
					    double** __restrict__ rho_e,
					    double** __restrict__ rho_fluxes,
					    double** __restrict__ rho_v1_fluxes,
					    double** __restrict__ rho_v2_fluxes,
					    double** __restrict__ rho_e_fluxes,
					    double const* __restrict__ normal,
					    double const* __restrict__ area,
					    int const* e_idx, int* rank,
					    t8_locidx_t* indices, int nb_edges) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= nb_edges) return;

  double face_surface = area[i];

  int l_rank  = rank[e_idx[2 * i]];
  int l_index = indices[e_idx[2 * i]];

  int r_rank  = rank[e_idx[2 * i + 1]];
  int r_index = indices[e_idx[2 * i + 1]];

  double nx = normal[2*i];
  double ny = normal[2*i+1];

  double rho_l    = rho[l_rank][l_index];
  double rho_vx_l = rho_v1[l_rank][l_index];
  double rho_vy_l = rho_v2[l_rank][l_index];
  double rho_e_l  = rho_e[l_rank][l_index];

  double rho_r    = rho[r_rank][r_index];
  double rho_vx_r = rho_v1[r_rank][r_index];
  double rho_vy_r = rho_v2[r_rank][r_index];
  double rho_e_r  = rho_e[r_rank][r_index];

  // rotate from (x,y) basis to local basis (n,t)
  double rho_v1_l =  nx*rho_vx_l + ny*rho_vy_l;
  double rho_v2_l = -ny*rho_vx_l + nx*rho_vy_l;

  double rho_v1_r =  nx*rho_vx_r + ny*rho_vy_r;
  double rho_v2_r = -ny*rho_vx_r + nx*rho_vy_r;

  double u_L[5] = {rho_l, rho_v1_l, rho_v2_l, 0.0, rho_e_l};
  double u_R[5] = {rho_r, rho_v1_r, rho_v2_r, 0.0, rho_e_r};

  double F_star[5];

  double RHat[5][5];
  double DHat[5];

  kepes_compute_diffusion_matrix(u_L,
				 u_R,
				 F_star,
				 RHat,
				 DHat);

  double kappa = 1.4;
  double kappaM1 = kappa - 1.0;

  double sRho_L = 1.0/u_L[0];
  double sRho_R = 1.0/u_R[0];

  double Vel_L[3] = {u_L[1]*sRho_L, u_L[2]*sRho_L, u_L[3]*sRho_L};
  double Vel_R[3] = {u_R[1]*sRho_R, u_R[2]*sRho_R, u_R[3]*sRho_R};

  double p_L = kappaM1*(u_L[4]-0.5*(u_L[1]*Vel_L[0] + u_L[2]*Vel_L[1] + u_L[3]*Vel_L[2]));
  double p_R = kappaM1*(u_R[4]-0.5*(u_R[1]*Vel_R[0] + u_R[2]*Vel_R[1] + u_R[3]*Vel_R[2]));

  double sL =  log(p_L) - kappa*log(u_L[0]);
  double sR =  log(p_R) - kappa*log(u_R[0]);

  double rho_pL = u_L[0]/p_L;
  double rho_pR = u_R[0]/p_R;

  double vL[5];
  double vR[5];
  double vJump[5];
  double diss[5];

  vL[0] =  (kappa-sL)/(kappaM1) - 0.5*rho_pL*(Vel_L[0]*Vel_L[0] + Vel_L[1]*Vel_L[1] + Vel_L[2]*Vel_L[2]);
  vR[0] =  (kappa-sR)/(kappaM1) - 0.5*rho_pR*(Vel_R[0]*Vel_R[0] + Vel_R[1]*Vel_R[1] + Vel_R[2]*Vel_R[2]);

  vL[1] = rho_pL*Vel_L[0];
  vR[1] = rho_pR*Vel_R[0];

  vL[2] = rho_pL*Vel_L[1];
  vR[2] = rho_pR*Vel_R[1];

  vL[3] = rho_pL*Vel_L[2];
  vR[3] = rho_pR*Vel_R[2];

  vR[4] = -rho_pR;
  vL[4] = -rho_pL;

  for (size_t k=0; k<5; k++) {
    vJump[k] = vR[k] - vL[k];
  }
  for (size_t k=0; k<5; k++) {
    diss[k]  = DHat[k]*(RHat[0][k]*vJump[0] + RHat[1][k]*vJump[1] + RHat[2][k]*vJump[2] + RHat[3][k]*vJump[3] + RHat[4][k]*vJump[4]);
  }

  double diss_[5];
  for (size_t k=0; k<5; k++)
    diss_[k] = RHat[k][0]*diss[0] + RHat[k][1]*diss[1] + RHat[k][2]*diss[2] + RHat[k][3]*diss[3] + RHat[k][4]*diss[4];

  // Compute entropy stable numerical flux
  double F[5];
  for (size_t k=0; k<5; k++)
    F[k] = F_star[k] - 0.5*diss_[k];

  double rho_flux    = face_surface*F[0];
  double rho_v1_flux = face_surface*F[1];
  double rho_v2_flux = face_surface*F[2];
  double rho_e_flux  = face_surface*F[4];

  // rotate back
  double rho_vx_flux = nx*rho_v1_flux - ny*rho_v2_flux;
  double rho_vy_flux = ny*rho_v1_flux + nx*rho_v2_flux;

  atomicAdd(&rho_fluxes[l_rank][l_index], -rho_flux);
  atomicAdd(&rho_fluxes[r_rank][r_index],  rho_flux);

  atomicAdd(&rho_v1_fluxes[l_rank][l_index], -rho_vx_flux);
  atomicAdd(&rho_v1_fluxes[r_rank][r_index],  rho_vx_flux);

  atomicAdd(&rho_v2_fluxes[l_rank][l_index], -rho_vy_flux);
  atomicAdd(&rho_v2_fluxes[r_rank][r_index],  rho_vy_flux);

  atomicAdd(&rho_e_fluxes[l_rank][l_index], -rho_e_flux);
  atomicAdd(&rho_e_fluxes[r_rank][r_index],  rho_e_flux);
}
